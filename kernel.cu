#include "hip/hip_runtime.h"
#include <cstdint>
#include "hip/hip_runtime.h"
#include ""
#include <chrono>
#include <ctime>
#include <stdio.h>
#include <pthread.h>
#include <stdlib.h>
#include "UCPClient.h"

#ifdef _WIN32
#include <Windows.h>
#include <VersionHelpers.h>
#elif __linux__
#include <sys/socket.h> 
#include <netdb.h>
#endif

#include <ctime>
#include "Log.h"
#include <sstream>
#include "Constants.h"

#if NVML
#include "nvml.h"
#endif

// #pragma comment(lib, "nvml.lib")
// #pragma comment(lib, "nvapi.lib")
// #pragma comment(lib, "nvapi64.lib")

void promptExit(int exitCode);
pthread_mutex_t stratum_sock_lock;
pthread_mutex_t stratum_log_lock;

#ifdef __INTELLISENSE__
#define __launch_bounds__(blocksize)
#endif
//#define ROTR64(x, n)  (((x) >> (n)) | ((x) << (64 - (n))))
#define ROTR(x,n) ROTR64(x,n)
#define MAX_GPUS 16
#define CUDA_SAFE_CALL(call)                                          \
do {                                                                  \
	hipError_t err = call;                                           \
	if (hipSuccess != err) {                                         \
		fprintf(stderr, "Cuda error in func '%s' at line %i : %s.\n", \
		         __FUNCTION__, __LINE__, hipGetErrorString(err) );   \
		promptExit(-1);                                           \
	}                                                                 \
} while (0)



__constant__ static uint64_t __align__(8) c_512[16];
__constant__ static uint64_t __align__(8) c_vblake[8];
__constant__ uint64_t headerIn[8];
static uint32_t *d_nonces[MAX_GPUS];
static uint64_t *dev_nonceStart[MAX_GPUS];


__host__
void veri_init(int thr_id)
{
	CUDA_SAFE_CALL(hipMalloc(&d_nonces[thr_id], 1 * sizeof(uint32_t)));
	CUDA_SAFE_CALL(hipMalloc(&dev_nonceStart[thr_id], 1 * sizeof(uint64_t)));
};
void veri_setBlock(void *blockf)
{
	
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(headerIn), blockf, 8 * sizeof(uint64_t), 0, hipMemcpyHostToDevice));
};

__device__ __forceinline__ uint64_t ROTR64_L(uint64_t value,
	const int offset) {
	uint2 result;
			asm("shf.r.wrap.b32 %0, %1, %2, %3;" :
		"=r"(result.x) : "r"(__double2loint(__longlong_as_double(value))),
			"r"(__double2hiint(__longlong_as_double(value))), "r"(offset));
		asm("shf.r.wrap.b32 %0, %1, %2, %3;" :
		"=r"(result.y) : "r"(__double2hiint(__longlong_as_double(value))),
			"r"(__double2loint(__longlong_as_double(value))), "r"(offset));
	
	return __double_as_longlong(__hiloint2double(result.y,
		result.x));
}
__device__ __forceinline__ uint64_t ROTR64_H(uint64_t value,
	const int offset) {
	uint2 result;

		asm("shf.r.wrap.b32 %0, %1, %2, %3;" :
		"=r"(result.x) : "r"(__double2hiint(__longlong_as_double(value))),
			"r"(__double2loint(__longlong_as_double(value))), "r"(offset));
		asm("shf.r.wrap.b32 %0, %1, %2, %3;" :
		"=r"(result.y) : "r"(__double2loint(__longlong_as_double(value))),
			"r"(__double2hiint(__longlong_as_double(value))), "r"(offset));

	return __double_as_longlong(__hiloint2double(result.y,
		result.x));
}
#define B2B_G(v,a,b,c,d,x,y,c1,c2) { \
	v[a] = v[a] + v[b] + (x ^ c1); \
	v[d] ^= v[a]; \
	v[d] = ROTR64_H(v[d], 60); \
	v[c] = v[c] + v[d]; \
	v[b] = ROTR64_H(v[b] ^ v[c], 43); \
	v[a] = v[a] + v[b] + (y ^ c2); \
	v[d] = ROTR64_L(v[d] ^ v[a], 5); \
	v[c] = v[c] + v[d]; \
	v[b] = ROTR64_L(v[b] ^ v[c], 18); \
	v[d] ^= (~v[a] & ~v[b] & ~v[c]) | (~v[a] & v[b] & v[c]) | (v[a] & ~v[b] & v[c])   | (v[a] & v[b] & ~v[c]); \
    v[d] ^= (~v[a] & ~v[b] & v[c]) | (~v[a] & v[b] & ~v[c]) | (v[a] & ~v[b] & ~v[c]) | (v[a] & v[b] & v[c]); \
}


void grindNonces(uint32_t startNonce, uint32_t *nonceResult, uint64_t *hashStart, const uint64_t *header, int dev_id);
__device__ __constant__
static const uint8_t c_sigma_big[16][16] = {
	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
	{ 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	{ 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	{ 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
	{ 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 },

	{ 12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 },
	{ 13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 },
	{ 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5 },
	{ 10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13 , 0 },

	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
	{ 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	{ 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	{ 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
	{ 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 }
};

static const uint64_t cpu_u512[16] =
{
	0xA51B6A89D489E800ULL, 0xD35B2E0E0B723800ULL,
	0xA47B39A2AE9F9000ULL, 0x0C0EFA33E77E6488ULL,
	0x4F452FEC309911EBULL, 0x3CFCC66F74E1022CULL,
	0x4606AD364DC879DDULL, 0xBBA055B53D47C800ULL,
	0x531655D90C59EB1BULL, 0xD1A00BA6DAE5B800ULL,
	0x2FE452DA9632463EULL, 0x98A7B5496226F800ULL,
	0xBAFCD004F92CA000ULL, 0x64A39957839525E7ULL,
	0xD859E6F081AAE000ULL, 0x63D980597B560E6BULL
};

static const uint64_t cpu_vBlake_iv[8] = {
	0x4BBF42C1F006AD9Dull, 0x5D11A8C3B5AEB12Eull,
	0xA64AB78DC2774652ull, 0xC67595724658F253ull,
	0xB8864E79CB891E56ull, 0x12ED593E29FB41A1ull,
	0xB1DA3AB63C60BAA8ull, 0x6D20E50C1F954DEDull
};


__device__ 
uint64_t vBlake2(const uint64_t h0, const uint64_t h1, const uint64_t h2, const uint64_t h3, const uint64_t h4, const uint64_t h5, const uint64_t h6, const uint64_t h7, const uint64_t* u512, const uint64_t* s_vBlake, const uint8_t((*sigma)[16]))
{
	uint64_t h[8];
    uint64_t v[16];
	uint64_t m[16] = { 0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0 };

	h[0]  = v[8]  =  s_vBlake[0];
	
	h[0] ^= (uint64_t)(0x01010000 ^ 0x18);
	v[0] = h[0];
	v[9]  = v[1] = s_vBlake[1];
	v[10] = v[2] = s_vBlake[2];
	h[3]  = v[11] = v[3] = s_vBlake[3];
	v[12] = v[4] = s_vBlake[4];
	v[13] = v[5] = s_vBlake[5];
	h[6]  = v[14] = v[6] = s_vBlake[6];
	v[15] = v[7] = s_vBlake[7];
	
	

	m[0] = h0;
	m[1] = h1;
	m[2] = h2;
	m[3] = h3;
	m[4] = h4;
	m[5] = h5;
	m[6] = h6;
	m[7] = h7;		
	//vblake512_compress(h, b, c_sigma_big, s_u512);
	v[12] ^= 64;
	v[14] ^= (uint64_t)(0xffffffffffffffffull);// (long)(-1);

	#pragma unroll 16
	for (int i = 0; i < 16; i++) {
		B2B_G(v, 0, 4, 8, 12, m[sigma[i][1]], m[sigma[i][0]],
			u512[sigma[i][1]], u512[sigma[i][0]]);

		B2B_G(v, 1, 5, 9, 13, m[sigma[i][3]], m[sigma[i][2]],
			u512[sigma[i][3]], u512[sigma[i][2]]);

		B2B_G(v, 2, 6, 10, 14, m[sigma[i][5]], m[sigma[i][4]],
			u512[sigma[i][5]], u512[sigma[i][4]]);

		B2B_G(v, 3, 7, 11, 15, m[sigma[i][7]], m[sigma[i][6]],
			u512[sigma[i][7]], u512[sigma[i][6]]);

		B2B_G(v, 0, 5, 10, 15, m[sigma[i][9]], m[sigma[i][8]],
			u512[sigma[i][9]], u512[sigma[i][8]]);

		B2B_G(v, 1, 6, 11, 12, m[sigma[i][11]], m[sigma[i][10]],
			u512[sigma[i][11]], u512[sigma[i][10]]);

		B2B_G(v, 2, 7, 8, 13, m[sigma[i][13]], m[sigma[i][12]],
			u512[sigma[i][13]], u512[sigma[i][12]]);

		B2B_G(v, 3, 4, 9, 14, m[sigma[i][15]], m[sigma[i][14]],
			u512[sigma[i][15]], u512[sigma[i][14]]);
	}

	h[0] ^= v[0] ^ v[8];

	h[3] ^= v[3] ^ v[11];

	h[6] ^= v[6] ^ v[14];

	h[0] ^= h[3] ^ h[6];  //copied from  the java
	return h[0];
}


#if CPU_SHARES
#define WORK_PER_THREAD 256
#else
#define WORK_PER_THREAD 1
#endif

#if HIGH_RESOURCE
#define DEFAULT_BLOCKSIZE 0x80000
#define DEFAULT_THREADS_PER_BLOCK 256
#else
#define DEFAULT_BLOCKSIZE 512
#define DEFAULT_THREADS_PER_BLOCK 512
#endif

int blocksize = DEFAULT_BLOCKSIZE;
int threadsPerBlock = DEFAULT_THREADS_PER_BLOCK;
int opt_n_threads = 0;
short device_map[MAX_GPUS] = { 0 };
int gpu_threads = 1;
int active_gpus;
char * device_name[MAX_GPUS];
long  device_sm[MAX_GPUS] = { 0 };
short device_mpcount[MAX_GPUS] = { 0 };
int init[MAX_GPUS] = { 0 };

bool verboseOutput = false;
struct mining_attr {
	int dev_id;
	string host;
	int port;
	string username;
	string password;

};

/*
* Kernel function to search a range of nonces for a solution falling under the macro-configured difficulty (CPU=2^24, GPU=2^32).
*/
__global__ void vblakeHasher(uint32_t startnonce, uint32_t *nonceOut, uint64_t *hashStartOut)
{
	// Generate a unique starting nonce for each thread that doesn't overlap with the work of any other thread
	uint32_t nonce = (blockDim.x * blockIdx.x + threadIdx.x) + startnonce;
	__shared__ uint64_t s_u512[16],s_vblake[8];

	if (threadIdx.x < 16U) s_u512[threadIdx.x] = c_512[threadIdx.x];
	if (threadIdx.x < 8U) s_vblake[threadIdx.x] = c_vblake[threadIdx.x];

	uint64_t nonceHeaderSection = headerIn[7];
	
	//for (unsigned int nonce = workStart; nonce < workStart + WORK_PER_THREAD; nonce++) {
		// Zero out nonce position and write new nonce to last 32 bits of prototype header
		nonceHeaderSection &= 0x00000000FFFFFFFFu;
		nonceHeaderSection |= (((uint64_t)nonce) << 32);

		uint64_t hashStart = vBlake2(headerIn[0], headerIn[1], headerIn[2], headerIn[3], headerIn[4], headerIn[5], headerIn[6], nonceHeaderSection, s_u512, s_vblake, c_sigma_big);

		if ((hashStart & 0x00000000FFFFFFFFu) == 0) {
			// Check that found solution is better than existing solution if one has already been found on this run of the kernel (always send back highest-quality work)
			if (hashStartOut[0] > hashStart || hashStartOut[0] == 0) {
				nonceOut[0] = nonce;
				hashStartOut[0] = hashStart;
			}

			// exit loop early
			//nonce = workStart + WORK_PER_THREAD;
		}
	//}
}
int cuda_num_devices()
{
	int version = 0, GPU_N = 0;
	hipError_t err = hipDriverGetVersion(&version);
	if (err != hipSuccess) {
		printf("Unable to query CUDA driver version! Is an nVidia driver installed?\n");
		exit(1);
	}

	if (version < CUDART_VERSION) {
		printf("Your system does not support CUDA %d.%d API!\n",
			CUDART_VERSION / 1000, (CUDART_VERSION % 1000) / 10);
		exit(1);
	}

	err = hipGetDeviceCount(&GPU_N);
	if (err != hipSuccess) {
		printf("Unable to query number of CUDA devices! Is an nVidia driver installed?\n");
		exit(1);
	}
	return GPU_N;
}
void promptExit(int exitCode)
{
	cout << "Exiting in 10 seconds..." << endl;
	std::this_thread::sleep_for(std::chrono::milliseconds(10000));
	exit(exitCode);
}

/**
* Takes the provided timestamp and places it in the header
*/
void embedTimestampInHeader(uint8_t *header, uint32_t timestamp)
{
	header[55] = (timestamp & 0x000000FF);
	header[54] = (timestamp & 0x0000FF00) >> 8;
	header[53] = (timestamp & 0x00FF0000) >> 16;
	header[52] = (timestamp & 0xFF000000) >> 24;
}

/**
* Returns a 64-byte header to attempt to mine with.
*/
void getWork(UCPClient& ucpClient, uint32_t timestamp, uint64_t *header)
{
	//uint64_t *header = new uint64_t[8];
	ucpClient.copyHeaderToHash((byte *)header);
	embedTimestampInHeader((uint8_t*)header, timestamp);
	//return header;
}

int deviceToUse = 0;

#if NVML
nvmlDevice_t device;
void readyNVML(int deviceIndex) {
	nvmlInit();
	nvmlDeviceGetHandleByIndex(deviceIndex, &device);
}
int getTemperature() {
	unsigned int temperature;
	nvmlDeviceGetTemperature(device, NVML_TEMPERATURE_GPU, &temperature);
	return temperature;
}

int getCoreClock() {
	unsigned int clock;
	nvmlDeviceGetClock(device, NVML_CLOCK_GRAPHICS, NVML_CLOCK_ID_CURRENT, &clock);
	return clock;
}

int getMemoryClock() {
	unsigned int memClock;
	nvmlDeviceGetClock(device, NVML_CLOCK_MEM, NVML_CLOCK_ID_CURRENT, &memClock);
	return memClock;
}
#else
void readyNVML(int deviceIndex) {
	// Do Nothing
}

int getTemperature() {
	return -1;
}

int getCoreClock() {
	return -1;
}

int getMemoryClock() {
	return -1;
}
#endif

#define SHARE_SUBMISSION_NO_RESPONSE_WARN_THRESHOLD 50

void vprintf(char* toprint) {
	if (verboseOutput) {
		printf(toprint);
	}
}
void cuda_devicenames()
{
	hipError_t err;
	int GPU_N;
	err = hipGetDeviceCount(&GPU_N);
	if (err != hipSuccess)
	{
		printf("Unable to query number of CUDA devices! Is an nVidia driver installed?");
		exit(1);
	}

	if (opt_n_threads)
		GPU_N = min(MAX_GPUS, opt_n_threads);
	for (int i = 0; i < GPU_N; i++)
	{
			int dev_id = device_map[i];
		hipDeviceProp_t props;
		hipGetDeviceProperties(&props, dev_id);

		device_sm[dev_id] = (props.major * 100 + props.minor * 10);
		device_mpcount[dev_id] = (short)props.multiProcessorCount;

		if (device_name[dev_id]) {
			free(device_name[dev_id]);
			device_name[dev_id] = NULL;
		}

			device_name[dev_id] = strdup(props.name);
	}
}
void printHelpAndExit() {
	printf("VeriBlock vBlake GPU CUDA Miner v1.0\n");
	printf("Required Arguments:\n");
	printf("-o <poolAddress>           The pool address to mine to in the format host:port\n");
	printf("-u <username>              The username (often an address) used at the pool\n");
	printf("Optional Arguments:\n");
	printf("-p <password>              The miner/worker password to use on the pool\n");
	printf("-d <deviceNum>             The ordinal of the device to use (default 0)\n");
	printf("-tpb <threadPerBlock>      The threads per block to use with the Blake kernel (default %d)\n", DEFAULT_THREADS_PER_BLOCK);
	printf("-bs <blockSize>            The blocksize to use with the vBlake kernel (default %d)\n", DEFAULT_BLOCKSIZE);
	printf("-l <enableLogging>         Whether to log to a file (default true)\n");
	printf("-v <enableVerboseOutput>   Whether to enable verbose output for debugging (default false)\n");
	printf("\n");
	printf("Example command line:\n");
	printf("VeriBlock-NodeCore-PoW-CUDA -u VHT36jJyoVFN7ap5Gu77Crua2BMv5j -o testnet-pool-gpu.veriblock.org:8501 -l false\n");
	promptExit(0);
}

#ifdef _WIN32
static WSADATA g_wsa_data;
#endif

char net_init(void)
{
#ifdef _WIN32
	return (WSAStartup(MAKEWORD(2, 2), &g_wsa_data) == NO_ERROR);
#elif __linux__
	return 1;
#endif
}

void net_deinit(void)
{
#ifdef _WIN32
	WSACleanup();
#endif
}
static bool substringsearch(const char *haystack, const char *needle, int &match)
{
	int hlen = (int)strlen(haystack);
	int nlen = (int)strlen(needle);
	for (int i = 0; i < hlen; ++i)
	{
		if (haystack[i] == ' ') continue;
		int j = 0, x = 0;
		while (j < nlen)
		{
			if (haystack[i + x] == ' ') { ++x; continue; }
			if (needle[j] == ' ') { ++j; continue; }
			if (needle[j] == '#') return ++match == needle[j + 1] - '0';
			if (tolower(haystack[i + x]) != tolower(needle[j])) break;
			++j; ++x;
		}
		if (j == nlen) return true;
	}
	return false;
}
int cuda_finddevice(char *name)
{
	int num = cuda_num_devices();
	int match = 0;
	for (int i = 0; i < num; ++i)
	{
		hipDeviceProp_t props;
		if (hipGetDeviceProperties(&props, i) == hipSuccess)
			if (substringsearch(props.name, name, match)) return i;
	}
	return -1;
}

string net_dns_resolve(const char* hostname)
{
	struct addrinfo hints, *results, *item;
	int status;
	char ipstr[INET6_ADDRSTRLEN];

	memset(&hints, 0, sizeof hints);
	hints.ai_family = AF_UNSPEC;  /* AF_INET6 to force version */
	hints.ai_socktype = SOCK_STREAM;

	if ((status = getaddrinfo(hostname, NULL, &hints, &results)) != 0)
	{
		fprintf(stderr, "failed to resolve hostname \"%s\": %s", hostname, gai_strerror(status));
		return "invalid hostname";
	}

	printf("IP addresses for %s:\n\n", hostname);

	string ret;

	for (item = results; item != NULL; item = item->ai_next)
	{
		void* addr;
		char* ipver;

		/* get pointer to the address itself */
		/* different fields in IPv4 and IPv6 */
		if (item->ai_family == AF_INET)  /* address is IPv4 */
		{
			struct sockaddr_in* ipv4 = (struct sockaddr_in*)item->ai_addr;
			addr = &(ipv4->sin_addr);
			ipver = "IPv4";
		}
		else  /* address is IPv6 */
		{
			struct sockaddr_in6* ipv6 = (struct sockaddr_in6*)item->ai_addr;
			addr = &(ipv6->sin6_addr);
			ipver = "IPv6";
		}

		/* convert IP to a string and print it */
		inet_ntop(item->ai_family, addr, ipstr, sizeof ipstr);
		printf("  %s: %s\n", ipver, ipstr);
		ret = ipstr;
	}

	freeaddrinfo(results);
	return ret;
}

char outputBuffer[8192];


void* miner_thread(void* arg){
	// Run initialization of device before beginning timer
	struct mining_attr *arg_Struct =
		(struct mining_attr*) arg;

	pthread_mutex_lock(&stratum_sock_lock);
	UCPClient ucpClient(arg_Struct->host, arg_Struct->port, arg_Struct->username, arg_Struct->password);

	byte target[24];
	ucpClient.copyMiningTarget(target);
	uint64_t header[8];
	
	getWork(ucpClient, (uint32_t)std::time(0),header);
	pthread_mutex_unlock(&stratum_sock_lock);

	pthread_mutex_lock(&stratum_log_lock);
	unsigned long long startTime = std::time(0);
	pthread_mutex_unlock(&stratum_log_lock);
	//mutex unlock
	
	uint32_t nonceResult[1] = { 0 };
	uint64_t hashStart[1] = { 0 };
	uint32_t startNonce = 0;
	unsigned long long hashes = 0;
	uint32_t count = 0;
	int numLines = 0;

	// Mining loop
	while (true) {
		vprintf("top of mining loop\n");
		count++;
		long timestamp = (long)std::time(0);
		//delete[] header;
		vprintf("Getting work...\n");

		pthread_mutex_lock(&stratum_sock_lock);
		getWork(ucpClient, timestamp, header);
		vprintf("Getting job id...\n");
		int jobId = ucpClient.getJobId();
		pthread_mutex_unlock(&stratum_sock_lock);

		count++;
		vprintf("Running kernel...\n");
		grindNonces(startNonce, nonceResult, hashStart, header, arg_Struct->dev_id);
		
		vprintf("Kernel finished...\n");
		
		//mutex lock
		pthread_mutex_lock(&stratum_log_lock);
		unsigned long long totalTime = std::time(0) - startTime;
		pthread_mutex_unlock(&stratum_log_lock);
		//todo mutex unlock
		hashes += (blocksize * threadsPerBlock * WORK_PER_THREAD);
		if ((uint64_t)startNonce +  (uint64_t)(blocksize * threadsPerBlock * WORK_PER_THREAD) < (uint64_t)0xffffffff) {
			startNonce += (blocksize * threadsPerBlock * WORK_PER_THREAD);
		}
		else
			startNonce = 0;

		double hashSpeed = (double)hashes;
		hashSpeed /= (totalTime * 1024 * 1024);

		if (count % 10 == 0) {
			//mutex lock
			pthread_mutex_lock(&stratum_sock_lock);

			int validShares = ucpClient.getValidShares();
			int invalidShares = ucpClient.getInvalidShares();
			int totalAccountedForShares = invalidShares + validShares;
			int totalSubmittedShares = ucpClient.getSentShares();
			int unaccountedForShares = totalSubmittedShares - totalAccountedForShares;
			pthread_mutex_unlock(&stratum_sock_lock);
			//mutex unlock
			double percentage = ((double)validShares) / totalAccountedForShares;
			percentage *= 100;
			// printf("[GPU #%d (%s)] : %f MH/second    valid shares: %d/%d/%d (%.3f%%)\n", deviceToUse, selectedDeviceName.c_str(), hashSpeed, validShares, totalAccountedForShares, totalSubmittedShares, percentage);

			printf("[GPU: %d %s] : %0.2f MH/s shares: %d/%d/%d (%.3f%%)\n", arg_Struct->dev_id, device_name[arg_Struct->dev_id], hashSpeed, validShares, totalAccountedForShares, totalSubmittedShares, percentage);
		}

		if (nonceResult[0] != 0x01000000 && nonceResult[0] != 0) {
			uint32_t nonce = *nonceResult;
			nonce = (((nonce & 0xFF000000) >> 24) | ((nonce & 0x00FF0000) >> 8) | ((nonce & 0x0000FF00) << 8) | ((nonce & 0x000000FF) << 24));
			
			pthread_mutex_lock(&stratum_sock_lock);
			ucpClient.submitWork(jobId, timestamp, nonce);
			pthread_mutex_unlock(&stratum_sock_lock);
			
			nonceResult[0] = 0;

			char line[100];

			// Hash coming from GPU is reversed
			uint64_t hashFlipped = 0;
			hashFlipped |= (hashStart[0] & 0x00000000000000FF) << 56;
			hashFlipped |= (hashStart[0] & 0x000000000000FF00) << 40;
			hashFlipped |= (hashStart[0] & 0x0000000000FF0000) << 24;
			hashFlipped |= (hashStart[0] & 0x00000000FF000000) << 8;
			hashFlipped |= (hashStart[0] & 0x000000FF00000000) >> 8;
			hashFlipped |= (hashStart[0] & 0x0000FF0000000000) >> 24;
			hashFlipped |= (hashStart[0] & 0x00FF000000000000) >> 40;
			hashFlipped |= (hashStart[0] & 0xFF00000000000000) >> 56;

#if CPU_SHARES 
			sprintf(line, "\t Share Found @ 2^24! {%#018llx} [nonce: %#08lx]", hashFlipped, nonce);
#else
			sprintf(line, "\t Share Found @ 2^32! {%#018llx} [nonce: %#08lx]", hashFlipped, nonce);
#endif

			cout << line << endl;
			vprintf("Logging\n");
			Log::info(line);
			vprintf("Done logging\n");
			vprintf("Made line\n");

			numLines++;

			// Uncomment these lines to get access to this data for display purposes
			/*
			long long extraNonce = ucpClient.getStartExtraNonce();
			int jobId = ucpClient.getJobId();
			int encodedDifficulty = ucpClient.getEncodedDifficulty();
			string previousBlockHashHex = ucpClient.getPreviousBlockHash();
			string merkleRoot = ucpClient.getMerkleRoot();
			*/

		}
		vprintf("About to restart loop...\n");
	}

	printf("Resetting device...\n");
	CUDA_SAFE_CALL(hipDeviceReset());
	
}
int main(int argc, char *argv[])
{
	// Check for help argument (only -h)
	for (int i = 1; i < argc; i++) {
		char* argument = argv[i];

		if (!strcmp(argument, "-h"))
		{
			printHelpAndExit();
		}
	}

	if (argc % 2 != 1) {
		sprintf(outputBuffer, "GPU miner must be provided valid argument pairs!");
		cerr << outputBuffer << endl;
		printHelpAndExit();
	}

	string hostAndPort = ""; //  "94.130.64.18:8501";
	string username = ""; // "VGX71bcRsEh4HZzhbA9Nj7GQNH5jGw";
	string password = "";

	if (argc > 1)
	{
		for (int i = 1; i < argc; i += 2)
		{
			char* argument = argv[i];
			printf("%s\n", argument);
			if (argument[0] == '-' && argument[1] == 'd')
			{

				int device_thr[MAX_GPUS] = { 0 };
				int ngpus = cuda_num_devices();
				char* pch = strtok(argv[i + 1], ",");
				opt_n_threads = 0;
				while (pch != NULL && opt_n_threads < MAX_GPUS) {
					if (pch[0] >= '0' && pch[0] <= '9' && strlen(pch) <= 2)
					{
						if (atoi(pch) < ngpus)
							device_map[opt_n_threads++] = atoi(pch);
						else {
							printf("Non-existant CUDA device #%d specified in -d option\n\n", atoi(pch));
							printHelpAndExit();
						}
					}
					else {
						int device = cuda_finddevice(pch);
						if (device >= 0 && device < ngpus)
							device_map[opt_n_threads++] = device;
						else {
							printf("Non-existant CUDA device '%s' specified in -d option\n\n", pch);
							printHelpAndExit();
						}
					}
					pch = strtok(NULL, ",");
				}
				// count threads per gpu
				for (int n = 0; n < opt_n_threads; n++) {
					int device = device_map[n];
					device_thr[device]++;
				}
				for (int n = 0; n < ngpus; n++) {
					gpu_threads = max(gpu_threads, device_thr[n]);
				}

				//  if (strlen(argv[i + 1]) == 2) {
				//  device num >= 10
				//	deviceToUse = (argv[i + 1][0] - 48) * 10 + (argv[i + 1][1] - 48);
				//  }
				//  else {
				//	deviceToUse = argv[i + 1][0] - 48;
				//  }
			}
			else if (!strcmp(argument, "-o"))
			{
				hostAndPort = string(argv[i + 1]);
			}
			else if (!strcmp(argument, "-u"))
			{
				username = string(argv[i + 1]);
			}
			else if (!strcmp(argument, "-p"))
			{
				password = string(argv[i + 1]);
			}
			else if (!strcmp(argument, "-tpb"))
			{
				threadsPerBlock = stoi(argv[i + 1]);
			}
			else if (!strcmp(argument, "-bs"))
			{
				blocksize = stoi(argv[i + 1]);
			}
			else if (!strcmp(argument, "-l"))
			{
				// to lower case conversion
				for (int j = 0; j < strlen(argv[i + 1]); j++)
				{
					argv[i + 1][j] = tolower(argv[i + 1][j]);
				}
				if (!strcmp(argv[i + 1], "true") || !strcmp(argv[i + 1], "t"))
				{
					Log::setEnabled(true);
				}
				else
				{
					Log::setEnabled(false);
				}
			}
			else if (!strcmp(argument, "-v"))
			{
				// to lower case conversion
				for (int j = 0; j < strlen(argv[i + 1]); j++)
				{
					argv[i + 1][j] = tolower(argv[i + 1][j]);
				}
				if (!strcmp(argv[i + 1], "true") || !strcmp(argv[i + 1], "t"))
				{
					verboseOutput = true;
				}
				else
				{
					verboseOutput = false;
				}
			}
		}
	}
	else {
		printHelpAndExit();
	}

	pthread_mutex_init(&stratum_sock_lock, NULL);
	pthread_mutex_init(&stratum_log_lock, NULL);


	if (HIGH_RESOURCE) {
		sprintf(outputBuffer, "Resource Utilization: HIGH");
		cerr << outputBuffer << endl;
		Log::info(outputBuffer);
	}
	else {
		sprintf(outputBuffer, "Resource Utilization: LOW");
		cerr << outputBuffer << endl;
		Log::info(outputBuffer);
	}


	if (CPU_SHARES) {
		sprintf(outputBuffer, "Share Type: CPU");
		cerr << outputBuffer << endl;
		Log::info(outputBuffer);
	}
	else {
		sprintf(outputBuffer, "Share Type: GPU");
		cerr << outputBuffer << endl;
		Log::info(outputBuffer);
	}

	if (BENCHMARK) {
		sprintf(outputBuffer, "Benchmark Mode: ENABLED");
		cerr << outputBuffer << endl;
		Log::info(outputBuffer);
	}
	else {
		sprintf(outputBuffer, "Benchmark Mode: DISABLED");
		cerr << outputBuffer << endl;
		Log::info(outputBuffer);
	}



#ifdef _WIN32
	HANDLE consoleHandle = GetStdHandle(STD_OUTPUT_HANDLE);
#else
#endif

	if (hostAndPort.compare("") == 0) {
		string error = "You must specify a host in the command line arguments! Example: \n-o 127.0.0.1:8501 or localhost:8501";
		cerr << error << endl;
		Log::error(error);
		promptExit(-1);
	}

	if (username.compare("") == 0) {
		string error = "You must specify a username in the command line arguments! Example: \n-u V5bLSbCqj9VzQR3MNANqL13YC2tUep";
		cerr << error << endl;
		Log::error(error);
		promptExit(-1);
	}

	string host = hostAndPort.substr(0, hostAndPort.find(":"));
	//GetHostByName
	net_init();
	host = net_dns_resolve(host.c_str());
	net_deinit();

	string portString = hostAndPort.substr(hostAndPort.find(":") + 1);

	// Ensure that port is numeric
	if (portString.find_first_not_of("1234567890") != string::npos) {
		string error = "You must specify a host in the command line arguments! Example: \n-o 127.0.0.1:8501 or localhost:8501";
		cerr << error << endl;
		Log::error(error);
		promptExit(-1);
	}

	int port = stoi(portString);

	sprintf(outputBuffer, "Attempting to mine to pool %s:%d with username %s and password %s...", host.c_str(), port, username.c_str(), password.c_str());
	cout << outputBuffer << endl;
	Log::info(outputBuffer);




	active_gpus = cuda_num_devices();
	if (active_gpus == 0) {
		printf("No CUDA devices found! terminating.\n");
		exit(1);
	}
	for (int i = 0; i < MAX_GPUS; i++) {
		device_map[i] = i % active_gpus;
		device_name[i] = NULL;
	}
	cuda_devicenames();


/*	for (int i = 0; i < opt_n_threads; i++) {
		hipSetDevice(device_map[i]);
		hipDeviceReset();
		hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
		hipFuncSetCacheConfig(reinterpret_cast<const void*>(vblakeHasher), hipFuncCachePreferL1);
		hipError_t e = hipGetLastError();
		sprintf(outputBuffer, "Last error: %s\n", hipGetErrorString(e));
		cout << outputBuffer << endl;
		Log::info(outputBuffer);

	}
*/

	pthread_t tids[MAX_GPUS];
	struct mining_attr m_args[MAX_GPUS];

	for (int i = 0; i < opt_n_threads; i++) {
		m_args[i].host = host;
		m_args[i].port = port;
		m_args[i].username = username;
		m_args[i].password = password;
		m_args[i].dev_id = device_map[i];

		pthread_attr_t attr;
		pthread_attr_init(&attr);
		pthread_create(&tids[i], &attr, miner_thread, &m_args[i]);
	}

	pthread_join(tids[0], NULL);

}
uint32_t lastNonceStart = 0;

// Grind Through vBlake nonces with the provided header, setting the resultant nonce and associated hash start if a high-difficulty solution is found
void grindNonces(uint32_t startnonce, uint32_t *nonceResult, uint64_t *hashStart, const uint64_t *header, int dev_id)
{
	// Select GPU to run on
	if (!init[dev_id])
	{
		CUDA_SAFE_CALL(hipSetDevice(dev_id));
		hipDeviceReset();
		hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
		hipFuncSetCacheConfig(reinterpret_cast<const void*>(vblakeHasher), hipFuncCachePreferL1);
	// Allocate GPU buffers for nonce result and header
		veri_init(dev_id);
		init[dev_id] = 1;
	}
	
	// Copy starting nonce to GPU
	
	veri_setBlock((void*)header);
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_512), cpu_u512, sizeof(cpu_u512), 0, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_vblake), cpu_vBlake_iv, sizeof(cpu_vBlake_iv), 0, hipMemcpyHostToDevice));
	hipMemset(d_nonces[dev_id], 0x00, 1 * sizeof(uint32_t));
	hipMemset(dev_nonceStart[dev_id], 0x00, 1 * sizeof(uint64_t));

	// Launch a kernel on the GPU with one thread for each element.
	vblakeHasher << < blocksize, threadsPerBlock >> >(startnonce, d_nonces[dev_id], dev_nonceStart[dev_id]);
	hipDeviceSynchronize();
	hipMemcpy(nonceResult, d_nonces[dev_id], 1 * sizeof(uint32_t), hipMemcpyDeviceToHost);
	hipMemcpy(hashStart, dev_nonceStart[dev_id], 1 * sizeof(uint64_t), hipMemcpyDeviceToHost);
};
